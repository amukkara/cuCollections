#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cub/block/block_run_length_decode.cuh"
#include "hipcub/hipcub.hpp"
#include "input_utils.hpp"
#include <cuco/trie.cuh>
#include <cuco/trie_ref.cuh>
#include <limits>

const uint32_t FIND_PATH_BLOCK_SIZE = 1024;

const uint32_t MAX_PATH_BUFFER_SIZE = 100 * 1000;  // 8192
const uint32_t MAX_FRONTIER_SIZE    = 100 * 1000;

const uint32_t TOPK_KEYS_PER_LEVEL = 256;
const bool CHECK_FIND_PATHS_RESULT = false;

const uint32_t num_topk_id = 100;
const uint32_t max_depth   = 16;
const uint32_t max_paths   = 100;

struct Edge {
  uint32_t node_id;
  float score;
};

struct Path {
  uint32_t node_id;
  uint32_t level_id;
};

const float score_sentinel       = -std::numeric_limits<float>::max();
const float node_score_threshold = -11;

struct State {
  uint32_t node_id;
  float score;
};

template <typename T>
class PathEnumeration {
 public:
  PathEnumeration();
  ~PathEnumeration() noexcept(false);

  void find_paths(const cuco::experimental::trie<T>* trie,
                  const uint32_t* keys,
                  const float* scores,
                  uint32_t max_depth,
                  uint32_t max_paths,
                  uint32_t stream_id);
  void sync_streams();

 private:
  uint32_t num_levels_;

  const uint32_t num_streams = 1;
  std::vector<hipStream_t> streams;

  State** frontiers;
  State** next_frontiers;

  uint32_t* num_paths_outs;

  Path** path_buffers;
  float** score_buffers;

  T** path_values;
  uint32_t** path_offsets;

  void** sort_paths_temp_storage;
  size_t sort_paths_temp_storage_bytes;

 private:
  void sort_paths(uint32_t stream_id) const;
  void check_result(uint32_t stream_id, uint32_t max_paths) const;
};

template <typename T>
PathEnumeration<T>::PathEnumeration()
{
  CUCO_CUDA_TRY(hipMalloc(&num_paths_outs, sizeof(uint32_t) * num_streams));

  frontiers      = (State**)malloc(sizeof(State*) * num_streams);
  next_frontiers = (State**)malloc(sizeof(State*) * num_streams);

  path_buffers  = (Path**)malloc(sizeof(Path*) * 2 * num_streams);
  score_buffers = (float**)malloc(sizeof(float*) * 2 * num_streams);

  path_values  = (T**)malloc(sizeof(T*) * num_streams);
  path_offsets = (uint32_t**)malloc(sizeof(uint32_t*) * num_streams);

  sort_paths_temp_storage = (void**)malloc(sizeof(void*) * num_streams);

  sort_paths_temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairsDescending(nullptr,
                                            sort_paths_temp_storage_bytes,
                                            score_buffers[0],
                                            score_buffers[1],
                                            path_buffers[0],
                                            path_buffers[1],
                                            MAX_PATH_BUFFER_SIZE);

  streams.resize(num_streams);
  for (size_t i = 0; i < num_streams; i++) {
    CUCO_CUDA_TRY(hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking));

    CUCO_CUDA_TRY(hipMalloc(&frontiers[i], sizeof(State) * MAX_FRONTIER_SIZE));
    CUCO_CUDA_TRY(hipMalloc(&next_frontiers[i], sizeof(State) * MAX_FRONTIER_SIZE));

    CUCO_CUDA_TRY(hipMalloc(&path_buffers[i], sizeof(Path) * MAX_PATH_BUFFER_SIZE));
    CUCO_CUDA_TRY(hipMalloc(&path_buffers[i + num_streams], sizeof(Path) * MAX_PATH_BUFFER_SIZE));
    CUCO_CUDA_TRY(hipMalloc(&score_buffers[i], sizeof(float) * MAX_PATH_BUFFER_SIZE));
    CUCO_CUDA_TRY(
      hipMalloc(&score_buffers[i + num_streams], sizeof(float) * MAX_PATH_BUFFER_SIZE));

    CUCO_CUDA_TRY(hipMalloc(&path_values[i], sizeof(T) * 1000 * 100));
    CUCO_CUDA_TRY(hipMalloc(&path_offsets[i], sizeof(uint32_t) * 1000));

    CUCO_CUDA_TRY(hipMalloc(&sort_paths_temp_storage[i], sort_paths_temp_storage_bytes));
  }
}

template <typename T>
PathEnumeration<T>::~PathEnumeration() noexcept(false)
{
  for (size_t i = 0; i < num_streams; i++) {
    CUCO_CUDA_TRY(hipFree(frontiers[i]));
    CUCO_CUDA_TRY(hipFree(next_frontiers[i]));
    CUCO_CUDA_TRY(hipFree(path_buffers[2 * i]));
    CUCO_CUDA_TRY(hipFree(path_buffers[2 * i + 1]));
    CUCO_CUDA_TRY(hipFree(score_buffers[2 * i]));
    CUCO_CUDA_TRY(hipFree(score_buffers[2 * i + 1]));
    CUCO_CUDA_TRY(hipFree(path_values[i]));
    CUCO_CUDA_TRY(hipFree(path_offsets[i]));
    CUCO_CUDA_TRY(hipFree(sort_paths_temp_storage[i]));
    CUCO_CUDA_TRY(hipStreamDestroy(streams[i]));
  }
  free(frontiers);
  free(next_frontiers);
  free(path_buffers);
  free(score_buffers);
  free(path_values);
  free(path_offsets);
  free(sort_paths_temp_storage);

  CUCO_CUDA_TRY(hipFree(num_paths_outs));
}

template <typename T>
void PathEnumeration<T>::sync_streams()
{
  for (auto& stream : streams) {
    CUCO_CUDA_TRY(hipStreamSynchronize(stream));
  }
}

template <typename T>
__global__ void __launch_bounds__(FIND_PATH_BLOCK_SIZE, 1)
  find_paths_kernel(const cuco::experimental::trie<T>* t,
                    const uint32_t* keys,
                    const float* scores,
                    State* frontier_,
                    State* next_frontier_,
                    uint32_t* num_paths_out,
                    Path* path_buffer,
                    float* score_buffer,
                    uint32_t max_depth,
                    uint32_t max_paths);

template <typename T>
void PathEnumeration<T>::find_paths(const cuco::experimental::trie<T>* trie,
                                    const uint32_t* keys,
                                    const float* scores,
                                    uint32_t max_depth,
                                    uint32_t max_paths,
                                    uint32_t stream_id)
{
  assert(stream_id < streams.size());
  auto& stream = streams[stream_id];

  num_levels_ = trie->num_levels();
  max_depth   = min(num_levels_ - 1, max_depth);

  find_paths_kernel<<<1, FIND_PATH_BLOCK_SIZE, 0, stream>>>(trie->device_ptr_,
                                                            keys,
                                                            scores,
                                                            frontiers[stream_id],
                                                            next_frontiers[stream_id],
                                                            num_paths_outs + stream_id,
                                                            path_buffers[stream_id],
                                                            score_buffers[stream_id],
                                                            max_depth,
                                                            max_paths);
  CUCO_CUDA_TRY(hipStreamSynchronize(stream));

  sort_paths(stream_id);
  CUCO_CUDA_TRY(hipStreamSynchronize(stream));

  generate_full_paths<<<1, min(max_paths, 1024), 0, stream>>>(trie->device_ptr_,
                                                              path_buffers[stream_id],
                                                              score_buffers[stream_id],
                                                              path_values[stream_id],
                                                              path_offsets[stream_id],
                                                              max_paths);
  CUCO_CUDA_TRY(hipStreamSynchronize(stream));

  if (CHECK_FIND_PATHS_RESULT) { check_result(stream_id, max_paths); }
}

__device__ float score_node(const uint32_t* keys, const float* scores, uint32_t label, bool& match);

template <typename BV>
__device__ uint32_t init_node_pos(const BV& louds, uint32_t& node_id)
{
  uint32_t node_pos = 0;
  if (node_id != 0) {
    node_pos = louds.select(node_id - 1) + 1;
    node_id  = node_pos - node_id;
  }
  return node_pos;
}

template <typename T>
__global__ void __launch_bounds__(FIND_PATH_BLOCK_SIZE, 1)
  find_paths_kernel(const cuco::experimental::trie<T>* t,
                    const uint32_t* keys,
                    const float* scores,
                    State* frontier_,
                    State* next_frontier_,
                    uint32_t* num_paths_out,
                    Path* path_buffer,
                    float* score_buffer,
                    uint32_t max_depth,
                    uint32_t max_paths)
{
  uint32_t offset = threadIdx.x;
  while (offset < MAX_PATH_BUFFER_SIZE) {
    score_buffer[offset] = score_sentinel;
    offset += blockDim.x;
  }

  State* frontier      = frontier_;
  State* next_frontier = next_frontier_;
  uint32_t cur_depth   = 1;

  __shared__ uint32_t frontier_size;
  __shared__ uint32_t next_frontier_index;
  __shared__ uint32_t path_buffer_index;
  __shared__ uint32_t level_keys[TOPK_KEYS_PER_LEVEL];

  if (threadIdx.x == 0) {
    next_frontier_index = 0;
    path_buffer_index   = 0;

    frontier[0]   = {0, 0.0};
    frontier_size = 1;
  }
  __syncthreads();

  while (cur_depth <= max_depth and frontier_size > 0) {
    const auto& level = t->d_levels_ptr_[cur_depth];
    const auto& louds = t->d_louds_refs_ptr_[cur_depth];
    const auto& outs  = t->d_outs_refs_ptr_[cur_depth];

    uint32_t num_iters      = (frontier_size - 1) / blockDim.x + 1;
    uint32_t frontier_index = threadIdx.x;
    for (uint32_t iter = 0; iter < num_iters; iter++) {
      Edge parent     = {0, 0};
      uint32_t degree = 0;

      if (frontier_index < frontier_size) {
        const auto s      = frontier[frontier_index];
        uint32_t node_id  = s.node_id;
        uint32_t node_pos = init_node_pos(louds, node_id);
        uint32_t pos_end  = louds.find_next_set(node_pos);
        uint32_t node_end = node_id + (pos_end - node_pos);

        parent = {node_id, s.score};
        degree = node_end - node_id;
      }

      auto process_edge = [&](uint32_t node_id, float score, uint32_t label, bool terminal_path) {
        bool valid_path;
        if (cur_depth == 1) {
          valid_path = label == keys[0];
          score += scores[0];
        } else {
          uint32_t offset = (cur_depth - 1) * TOPK_KEYS_PER_LEVEL;
          score += score_node(level_keys, scores + offset, label, valid_path);
        }

        if (valid_path) {
          uint32_t insert_index = min(MAX_FRONTIER_SIZE - 1, atomicAdd(&next_frontier_index, 1));
          next_frontier[insert_index] = {node_id, score};

          if (terminal_path) {
            uint32_t insert_index = min(MAX_PATH_BUFFER_SIZE - 1, atomicAdd(&path_buffer_index, 1));
            path_buffer[insert_index]  = {node_id, cur_depth};
            score_buffer[insert_index] = score;
          }
        }
      };

      using RunItemT                              = Edge;
      using RunLengthT                            = uint32_t;
      constexpr uint32_t BLOCK_DIM_X              = FIND_PATH_BLOCK_SIZE;
      constexpr uint32_t RUNS_PER_THREAD          = 1;
      constexpr uint32_t DECODED_ITEMS_PER_THREAD = 3;

      using BlockRunLengthDecodeT =
        hipcub::BlockRunLengthDecode<RunItemT, BLOCK_DIM_X, RUNS_PER_THREAD, DECODED_ITEMS_PER_THREAD>;
      __shared__ typename BlockRunLengthDecodeT::TempStorage temp_storage;

      RunItemT run_values[RUNS_PER_THREAD];
      RunLengthT run_lengths[RUNS_PER_THREAD];
      run_values[0]  = parent;
      run_lengths[0] = degree;

      uint32_t total_decoded_size = 0;
      BlockRunLengthDecodeT block_rld(temp_storage, run_values, run_lengths, total_decoded_size);

      uint32_t decoded_window_offset = 0U;
      while (decoded_window_offset < total_decoded_size) {
        RunItemT decoded_items[DECODED_ITEMS_PER_THREAD];
        RunLengthT relative_offsets[DECODED_ITEMS_PER_THREAD];

        int num_valid_items = total_decoded_size - decoded_window_offset;
        block_rld.RunLengthDecode(decoded_items, relative_offsets, decoded_window_offset);
        decoded_window_offset += BLOCK_DIM_X * DECODED_ITEMS_PER_THREAD;

        uint32_t labels[DECODED_ITEMS_PER_THREAD];
        bool terminal_paths[DECODED_ITEMS_PER_THREAD];

        uint32_t start_offset       = DECODED_ITEMS_PER_THREAD * threadIdx.x;
        uint32_t thread_valid_items = 0;
        // Manual loop unrolling from 0 to DECODED_ITEMS_PER_THREAD - 1
        thread_valid_items += start_offset + 0 < num_valid_items;
        thread_valid_items += start_offset + 1 < num_valid_items;
        thread_valid_items += start_offset + 2 < num_valid_items;

        for (uint32_t item = 0; item < thread_valid_items; item++) {
          auto node_id         = decoded_items[item].node_id + relative_offsets[item];
          labels[item]         = level.d_labels_ptr[node_id];
          terminal_paths[item] = outs.get(node_id);
        }

        for (uint32_t item = 0; item < thread_valid_items; item++) {
          auto node_id = decoded_items[item].node_id + relative_offsets[item];
          process_edge(node_id, decoded_items[item].score, labels[item], terminal_paths[item]);
        }
      }

      frontier_index += blockDim.x;
      __syncthreads();
    }  // Iters end

    State* temp   = frontier;
    frontier      = next_frontier;
    next_frontier = temp;

    cur_depth++;

    if (cur_depth <= max_depth) {
      uint32_t offset = (cur_depth - 1) * TOPK_KEYS_PER_LEVEL;
      for (uint32_t pos = threadIdx.x; pos < TOPK_KEYS_PER_LEVEL; pos += blockDim.x) {
        level_keys[pos] = keys[offset + pos];
      }
    }

    if (threadIdx.x == 0) {
      frontier_size       = min(MAX_FRONTIER_SIZE, next_frontier_index);
      next_frontier_index = 0;
    }

    __syncthreads();
  }  // Level end

  if (threadIdx.x == 0) { *num_paths_out = path_buffer_index; }
}

template <typename T>
void PathEnumeration<T>::sort_paths(uint32_t stream_id) const
{
  auto& stream            = streams[stream_id];
  auto temp_storage_bytes = sort_paths_temp_storage_bytes;
  hipcub::DeviceRadixSort::SortPairsDescending(sort_paths_temp_storage[stream_id],
                                            temp_storage_bytes,
                                            score_buffers[stream_id],
                                            score_buffers[num_streams + stream_id],
                                            path_buffers[stream_id],
                                            path_buffers[num_streams + stream_id],
                                            MAX_PATH_BUFFER_SIZE,
                                            0,
                                            sizeof(float) * 8,
                                            stream);
}

template <typename T>
void PathEnumeration<T>::check_result(uint32_t stream_id, uint32_t max_paths) const
{
  auto& stream = streams[stream_id];

  uint32_t num_paths = 0;
  hipMemcpyAsync(
    &num_paths, num_paths_outs + stream_id, sizeof(uint32_t), hipMemcpyDeviceToHost, stream);
  CUCO_CUDA_TRY(hipStreamSynchronize(stream));
  std::cout << "Num paths " << num_paths << std::endl;

  std::vector<Path> paths_out(max_paths);
  std::vector<float> scores_out(max_paths);
  hipMemcpyAsync(&paths_out[0],
                  path_buffers[num_streams + stream_id],
                  sizeof(Path) * min(max_paths, MAX_PATH_BUFFER_SIZE),
                  hipMemcpyDeviceToHost,
                  stream);
  hipMemcpyAsync(&scores_out[0],
                  score_buffers[num_streams + stream_id],
                  sizeof(float) * min(max_paths, MAX_PATH_BUFFER_SIZE),
                  hipMemcpyDeviceToHost,
                  stream);

  CUCO_CUDA_TRY(hipStreamSynchronize(stream));

  for (uint32_t path_id = 0; path_id < 5; path_id++) {
    auto path = paths_out[path_id];
    std::cout << "Path " << path_id << ": " << path.node_id << " @ " << path.level_id << " "
              << scores_out[path_id] << std::endl;
  }
}

template <typename T>
__device__ void backtrace_path(const cuco::experimental::trie<T>* t,
                               int32_t level_id,
                               uint32_t start_node_id,
                               int32_t buffer_pos,
                               T* buffer)
{
  if (level_id == 0) { return; }
  uint32_t node_pos = t->d_louds_refs_ptr_[level_id].select0(start_node_id);
  for (; level_id >= 1; level_id--) {
    const auto& level    = t->d_levels_ptr_[level_id];
    uint32_t rank        = t->d_louds_refs_ptr_[level_id].rank(node_pos);
    uint32_t node_id     = node_pos - rank;
    buffer[--buffer_pos] = level.d_labels_ptr[node_id];  // insert in reverse order

    if (level_id > 1) { node_pos = t->d_louds_refs_ptr_[level_id - 1].select0(rank); }
  }
}

template <typename T>
__global__ void generate_full_paths(const cuco::experimental::trie<T>* t,
                                    Path* path_buffer,
                                    float* score_buffer,
                                    T* path_values,
                                    uint32_t* path_offsets,
                                    uint32_t num_paths)
{
  if (threadIdx.x < num_paths) { path_offsets[threadIdx.x] = path_buffer[threadIdx.x].level_id; }
  __syncthreads();
  if (threadIdx.x == 0) {
    for (uint32_t id = 1; id < num_paths + 1; id++) {
      path_offsets[id] = path_offsets[id - 1] + path_offsets[id];
    }
  }
  __syncthreads();

  if (threadIdx.x < num_paths) {
    const Path p    = path_buffer[threadIdx.x];
    uint32_t offset = path_offsets[threadIdx.x + 1];
    backtrace_path(t, p.level_id, p.node_id, offset, path_values);
  }
}

__device__ __forceinline__ float score_node(const uint32_t* keys,
                                            const float* scores,
                                            uint32_t label,
                                            bool& match)
{
  uint32_t ret = (keys[128] <= label) * 128;
  ret += (keys[ret + 64] <= label) * 64;
  ret += (keys[ret + 32] <= label) * 32;
  ret += (keys[ret + 16] <= label) * 16;
  ret += (keys[ret + 8] <= label) * 8;
  ret += (keys[ret + 4] <= label) * 4;
  ret += (keys[ret + 2] <= label) * 2;
  ret += (keys[ret + 1] <= label) * 1;

  float val = scores[ret];
  match     = keys[ret] == label and val > node_score_threshold;
  return match * val + (1 - match) * score_sentinel;
}

int main(void)
{
  using KeyType = uint32_t;
  cuco::experimental::trie<KeyType> trie;

  for (auto key : read_dataset(10 * 1000 * 1000)) {
    trie.insert(key);
  }
  trie.build();

  std::cout << "#keys " << trie.n_keys() << "   "
            << "#nodes " << trie.n_nodes() << std::endl;

  vector<const uint32_t*> topk_keys;
  vector<const float*> topk_scores;
  read_topk_keys_and_scores(topk_keys, topk_scores, num_topk_id, max_depth);

  PathEnumeration<KeyType> pe;
  for (size_t topk_id = 0; topk_id < num_topk_id; topk_id++) {  // warmup
    pe.find_paths(&trie, topk_keys[topk_id], topk_scores[topk_id], max_depth, max_paths, 0);
  }
  pe.sync_streams();

  auto begin            = high_resolution_clock::now();
  uint32_t topk_repeats = 10;
  for (uint32_t repeat = 0; repeat < topk_repeats; repeat++) {
    //#pragma omp parallel for num_threads(num_streams)
    for (size_t topk_id = 0; topk_id < num_topk_id; topk_id++) {
      pe.find_paths(&trie, topk_keys[topk_id], topk_scores[topk_id], max_depth, max_paths, 0);
    }
    pe.sync_streams();
  }
  uint32_t total_calls = topk_repeats * num_topk_id;
  cout << "GPU find paths " << get_milliseconds(begin) / total_calls << " msec" << endl;

  return 0;
}
